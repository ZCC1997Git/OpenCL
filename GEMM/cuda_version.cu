#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// 定义矩阵大小
#define M 1024*8
#define N 1024*8
#define K 1024*8

// 定义计时器
float milliseconds = 0;

// CUDA错误检查宏
#define CUDA_CHECK(call) \
do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", \
                __FILE__, __LINE__, hipGetErrorString(error)); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

int main() {
    // 分配内存和初始化数据
    float *h_A, *h_B, *h_C;
    h_A = new float[M * K];
    h_B = new float[K * N];
    h_C = new float[M * N];

    for (int i = 0; i < M * K; ++i)
        h_A[i] = i;

    for (int i = 0; i < K * N; ++i)
        h_B[i] = i;

    // 初始化CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 在设备上分配内存
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, M * K * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_B, K * N * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_C, M * N * sizeof(float)));

    // 将数据从主机复制到设备
    CUDA_CHECK(hipMemcpy(d_A, h_A, M * K * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K * N * sizeof(float), hipMemcpyHostToDevice));

    // 创建CUDA事件
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    // 启动计时器
    CUDA_CHECK(hipEventRecord(start));

    // 执行矩阵乘法
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_A, M, d_B, K, &beta, d_C, M);

    // 停止计时器
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    // 将结果从设备复制回主机
    CUDA_CHECK(hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // 打印结果
    //std::cout << "Result Matrix:" << std::endl;
    //for (int i = 0; i < M; ++i) {
     //   for (int j = 0; j < N; ++j) {
    //        std::cout << h_C[i * N + j] << " ";
    //    }
    //    std::cout << std::endl;
   // }

    // 打印计时结果
    std::cout << "Time elapsed: " << milliseconds << " ms" << std::endl;

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    hipblasDestroy(handle);
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}
      
